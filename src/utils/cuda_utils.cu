#include "utils/cuda_utils.h"

void check_cuda_error(hipError_t err, const char* const func, const char* const file, const int line) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\"\n", file, line, 
                static_cast<unsigned int>(err), hipGetErrorString(err), func);
        exit(EXIT_FAILURE);
    }
}